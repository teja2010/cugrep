/* file that implements the comparing */
#include <stdio.h>
#include <stdint.h>
#include "common.h"
#include <vector>
#include <algorithm>
#include <stack>
#include <hip/hip_runtime_api.h>



//struct nfa_state {
//	union {
//		uint32_t ignore_this;
//		struct {
//			uint8_t curr_state;
//			uint8_t match_char;
//			uint8_t next_state;
//			uint8_t again_ignore;
//		};
//	};
//};


/* NFA (non-deterministic finite automata) to represent a limited regex expression.
 * supports:
 *	"e*"  : matches 0 or more of the preceding expression e
 *	"e+"  : matches 0 or more of the preceding expression e
 *	"e|f" : matches expression e or expression f
 *
 *   Future work: Add support for:
 *	"."   : match any character TODO
 *	"(e)" : enclose a multi-character expression e
 *
 * returns nfa_blk_p 's length on success
 *         -1 on failure
 */
int build_nfa(char *regex, int regex_len, uint8_t **nfa_blk_p) {

	uint8_t *nfa_blk = NULL;

	nfa_blk = (uint8_t*)calloc(400, sizeof(uint8_t));
	*nfa_blk_p = nfa_blk;

	int idx = 0;

	int nfa_idx = 0;
	int state_counter = 0;
	std::stack<int> expression_start = {};
	std::vector<int> expression_end = {};

	while(idx < regex_len) {

		// check validity of character
		// i.e. (a,z) || (A,Z) || {*, +, ?, (, ) }
		if (!((regex[idx] >= 'a' && regex[idx] <= 'z') ||
		      (regex[idx] >= 'A' && regex[idx] <= 'Z') ||
		      regex[idx] == '*' || regex[idx] == '+' ||
		      regex[idx] == '?' || regex[idx] == '|'  )) {
		      //regex[idx] == '(' || regex[idx] == ')')) {
			printf("Invalid character %c(%d)\n",
					regex[idx], regex[idx]);
			return -1;
		}

		switch(regex[idx]) {
		case '*': {
			if (idx == 0) {
				printf("Invalid regex, * at begining\n");
				return -1;
			}

			int cs, mc, ns;
			cs = NFA_CURR_STATE(nfa_blk, nfa_idx-1);
			mc = NFA_MATCH_CHAR(nfa_blk, nfa_idx-1);
			ns = cs;
			NFA_SET(nfa_blk, nfa_idx-1, cs, mc, ns);
			state_counter--;
		}
		break;

		case '+': {
			if (idx == 0) {
				printf("Invalid regex, + at begining\n");
				return -1;
			}

			int cs, mc, ns;
			cs = state_counter;
			mc = NFA_MATCH_CHAR(nfa_blk, nfa_idx-1);
			ns = cs;
			NFA_SET(nfa_blk, nfa_idx, cs, mc, ns);
			nfa_idx++;

		}
		break;

		case '|': {
			if (idx == 0) {
				printf("Invalid regex, | at begining\n");
				return -1;
			}
			expression_end.push_back(state_counter);
		}
		break;

		default: {
			int curr_state = state_counter++;
			if (expression_start.size() == 0) {
				expression_start.push(curr_state);
			}

			if (idx > 0 && regex[idx-1] == '|') {
				curr_state = expression_start.top();
			}

			NFA_SET(nfa_blk, nfa_idx, curr_state, regex[idx], state_counter);
			nfa_idx++;
		}
		}

		idx++;
	}
	expression_end.push_back(state_counter);

	std::vector<uint32_t> temp_nfa = {};
	for(int i=0; i < nfa_idx ; i++) {
		uint32_t bb = ((uint32_t)NFA_CURR_STATE(nfa_blk, i) << 16) +
		              ((uint32_t)NFA_NEXT_STATE(nfa_blk, i) << 8) +
		              ((uint32_t)NFA_MATCH_CHAR(nfa_blk, i)     );
		int next_state = NFA_NEXT_STATE(nfa_blk, i);

		for (int st : expression_end) {
			if (next_state == st) {
				bb |= 0xff00;
				break;
			}
		}

		//printf("bb : %lx\n", bb);
		temp_nfa.push_back(bb);
	}
	std::sort(temp_nfa.begin(), temp_nfa.end());

	for(uint i =0; i < temp_nfa.size(); i++) {
		uint32_t tn = temp_nfa[i];
		//printf("tn : %lx\n", tn);
		NFA_SET(nfa_blk, i, (tn >> 16) & 0xff ,
		                    (tn) & 0xff , (tn >> 8) & 0xff);
	}

	//printf("NFA_BLK:\n");
	//for(uint i =0; i < temp_nfa.size(); i++) {
	//	printf(": %d %c %d\n",  NFA_CURR_STATE(nfa_blk, i),
	//				NFA_MATCH_CHAR(nfa_blk, i),
	//				NFA_NEXT_STATE(nfa_blk, i));
	//}

	return nfa_idx;
}

// the function logic that will be used to match lines in the kernel.
bool match(uint8_t *nfa, int nfa_len, char* str, int slen)
{
	int thread_idx = 0;

	int idx = 0;
	int nfa_idx = 0;
	int state = 0;
	bool reset = true;


	while(str[idx] !='\0') {
		reset = true;
		for(int i=nfa_idx; i< nfa_len; i++) {
			if(NFA_CURR_STATE(nfa, i) != state)
				break;

			if(NFA_MATCH_CHAR(nfa, i) == str[idx] ) {
				state = NFA_NEXT_STATE(nfa, i);
				reset = false;
				break;
			}
		}

		// nothing matched, reset state
		if (reset) {
			idx = idx - state;
			state = 0;
			nfa_idx = 0;
		}

		if (state == 0xff) {
			return true;
		}

		while(nfa_idx < nfa_len && NFA_CURR_STATE(nfa, nfa_idx) < state) {
			nfa_idx++;
		}

		idx++;
	}

	return false;
}
